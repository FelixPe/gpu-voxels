// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// This file is part of the GPU Voxels Software Library.
//
// This program is free software licensed under the CDDL
// (COMMON DEVELOPMENT AND DISTRIBUTION LICENSE Version 1.0).
// You can find a copy of this license in LICENSE.txt in the top
// directory of the source code.
//
// © Copyright 2014 FZI Forschungszentrum Informatik, Karlsruhe, Germany
//
// -- END LICENSE BLOCK ------------------------------------------------

//----------------------------------------------------------------------
/*!\file
 *
 * \author  Andreas Hermann
 * \date    2015-03-11
 *
 *
 */
//----------------------------------------------------------------------



#include "gpu_voxels/robot/urdf_robot/robot_to_gpu.h"
#include <gpu_voxels/helpers/cuda_datatypes.h>
#include "gpu_voxels/robot/kernels/KinematicOperations.h"

namespace gpu_voxels {
namespace robot {


RobotToGPU::RobotToGPU(Robot* _robot) :
  m_robot(_robot)
{
  // allocate a copy of the pointcloud, which will hold the transformed version
  m_link_pointclouds_transformed = new MetaPointCloud( m_robot->getLinkPointclouds());

  HANDLE_CUDA_ERROR(hipMalloc((void** )&m_transformation_dev, sizeof(Matrix4f)));

}

RobotToGPU::~RobotToGPU()
{
  HANDLE_CUDA_ERROR(hipFree(m_transformation_dev));
}

void RobotToGPU::setConfiguration(const std::map<std::string, float> &jointmap)
{
  // first update the joints of the URDF model.
  m_robot->setConfiguration(jointmap);


  // iterate over all joints that own a pointcloud.
  for(uint16_t i = 0; i < m_robot->getLinkPointclouds()->getNumberOfPointclouds(); i++)
  {
    // get the trafo of the according URDF link
    m_transformation = m_robot->getLink( m_robot->getLinkPointclouds()->getCloudName(i) )->getPoseAsGpuMat4f();
//    std::cout << "RobotToGPU::update() transform of " << robot->getLinkPointclouds()->getCloudName(i)
//              << " = " << transformation << std::endl;

    HANDLE_CUDA_ERROR(
        hipMemcpy(m_transformation_dev, &m_transformation, sizeof(Matrix4f), hipMemcpyHostToDevice));

    m_math.computeLinearLoad(m_link_pointclouds_transformed->getPointcloudSize(i),
                             &m_blocks, &m_threads_per_block);
    hipDeviceSynchronize();
    // transform the cloud via Kernel.
    kernelKinematicChainTransform<<< m_blocks, m_threads_per_block >>>
       (i, m_transformation_dev,
        m_robot->getLinkPointclouds()->getDeviceConstPointer(),
        m_link_pointclouds_transformed->getDevicePointer());

    HANDLE_CUDA_ERROR(hipDeviceSynchronize());
  }
}

void RobotToGPU::getConfiguration(const std::map<std::string, float> jointmap)
{
  m_robot->getConfiguration(jointmap);
}

const MetaPointCloud* RobotToGPU::getTransformedClouds()
{
  return m_link_pointclouds_transformed;
}

} // namespace robot
} // namespace gpu_voxels
